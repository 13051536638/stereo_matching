#include "hip/hip_runtime.h"
#include "../gpu_inc/cost.cuh"


__global__ void cu_Build_cost_table(uchar *d_ll, uchar *d_rr,
																   uint64_t *d_cost_table_l, 
	                                                               uint64_t *d_cost_table_r,
	                                                               int img_w, int img_h,
																   int win_w, int win_h)
{
	int index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (index > img_w * img_h - 1)  return;
	int col = index % img_w;
	int row = index / img_w;

	uint64_t value_l = 0, value_r = 0;
	uchar ctr_pixel_l = d_ll[row*img_w + col];
	uchar ctr_pixel_r = d_rr[row*img_w + col];

	for (int i = -win_h / 2; i <= win_h / 2; i++)
	{
		int y = MAX(row + i, 0);		// check border
		y = MIN(y, img_h - 1);
		for (int j = -win_w / 2; j <= win_w / 2; j++)
		{
			if (i == 0 && j == 0)
				continue;
			int x = MAX(col + j, 0);
			x = MIN(x, img_w - 1);
			int index_ = y * img_w + x;
			value_l = (value_l | (d_ll[index_] > ctr_pixel_l)) << 1;
			value_r = (value_r | (d_rr[index_] > ctr_pixel_r)) << 1;
		}
	}
	d_cost_table_l[row*img_w + col] = value_l;
	d_cost_table_r[row*img_w + col] = value_r;
	return;
}


__global__ void cu_Build_dsi_from_table(uint64_t *d_cost_table_l,
																		   uint64_t *d_cost_table_r,
																		   float *d_cost,
																		   int img_w, int img_h, int max_disp)
{
	int index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (index > img_w * img_h - 1)  return;
	int col = index % img_w;
	int row = index / img_w;

	for (int d = 0; d < max_disp; d++)
	{
		int dst_index = row * img_w * max_disp + col * max_disp + d;
		uint64_t ct_l = d_cost_table_l[row*img_w + col];
		uint64_t ct_r = d_cost_table_r[row*img_w + MAX(col - d, 0)];
		d_cost[dst_index] = cu_hamming_cost(ct_l, ct_r);
	}
}


__device__ int cu_hamming_cost(uint64_t ct_l, uint64_t ct_r)
{
	uint64_t not_the_same = ct_l ^ ct_r;
	// find the number of '1', log(N)
	int cnt = 0;
	while (not_the_same)
	{
		//std::cout << not_the_same << std::endl;
		cnt += (not_the_same & 1);
		not_the_same >>= 1;
	}
	return cnt;
}