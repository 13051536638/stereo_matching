#include "../gpu_inc/SGM.cuh"


GPU_SGM::GPU_SGM()
{
	hipSetDevice(0);
	hipMalloc((void**)&d_ll, IMG_H* IMG_W * sizeof(uchar));
	hipMalloc((void**)&d_rr, IMG_H * IMG_W * sizeof(uchar));
	hipMalloc((void**)&d_disp, IMG_H * IMG_W * sizeof(uchar));	
	hipMalloc((void**)&d_cost_table_l, IMG_H * IMG_W * CU_MAX_DISP * sizeof(uint64_t));
	hipMalloc((void**)&d_cost_table_r, IMG_H * IMG_W * CU_MAX_DISP * sizeof(uint64_t));
	hipMalloc((void**)&d_cost, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));

	hipMalloc((void**)&d_L1, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_L2, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_L3, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_L4, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_min_L1, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_min_L1, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_min_L1, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	hipMalloc((void**)&d_min_L1, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	if (CU_USE_8_PATH)
	{
		hipMalloc((void**)&d_L5, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_L6, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_L7, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_L8, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_min_L5, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_min_L6, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_min_L7, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
		hipMalloc((void**)&d_min_L8, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float));
	}

	P1 = 10;
	P2 = 100;
}



GPU_SGM::~GPU_SGM()
{
	hipFree(d_ll);
	hipFree(d_rr);
	hipFree(d_disp);
	hipFree(d_cost_table_l);
	hipFree(d_cost_table_r);
	hipFree(d_cost);

	hipFree(d_L1);
	hipFree(d_L2);
	hipFree(d_L3);
	hipFree(d_L4);
	hipFree(d_min_L1);
	hipFree(d_min_L2);
	hipFree(d_min_L3);
	hipFree(d_min_L4);
	if (CU_USE_8_PATH)
	{
		hipFree(d_L5);
		hipFree(d_L6);
		hipFree(d_L7);
		hipFree(d_L8);
		hipFree(d_min_L5);
		hipFree(d_min_L6);
		hipFree(d_min_L7);
		hipFree(d_min_L8);
	}
}


void GPU_SGM::Process(Mat &ll, Mat &rr, Mat &disp, float *cost)
{
	hipSetDevice(0);
	hipMemcpy(d_ll, ll.data, IMG_H* IMG_W * sizeof(uchar), hipMemcpyHostToDevice);
	hipMemcpy(d_rr, rr.data, IMG_H* IMG_W * sizeof(uchar), hipMemcpyHostToDevice);

	dim3 grid, block;
	grid.x = (IMG_W - 1) / 32 + 1;
	grid.y = (IMG_H - 1) / 32 + 1;
	block.x = 32;
	block.y = 32;

	cu_Build_cost_table<<<grid, block>>>(d_ll, d_rr, 
		                                                                 d_cost_table_l, 
		                                                                 d_cost_table_r, 
		                                                                 IMG_W, IMG_H, 
		                                                                 CU_WIN_W, CU_WIN_H);
	printf("cost table finished\n");
	cu_Build_dsi_from_table<<<grid, block>>>(d_cost_table_l, 
		                                                                          d_cost_table_r, 
		                                                                          d_cost,
		                                                                          IMG_W, IMG_H, CU_MAX_DISP);
	hipDeviceSynchronize();
	printf("dsi finished\n");

	hipMemcpy(cost, d_cost, IMG_H * IMG_W * CU_MAX_DISP * sizeof(float), hipMemcpyDeviceToHost);
}